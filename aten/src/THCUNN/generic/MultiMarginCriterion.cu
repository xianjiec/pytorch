
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiMarginCriterion.cu"
#else

// TODO: improve error messages
void THNN_(MultiMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           int64_t reduction,
           int p,
           THCTensor *weights,
           accreal margin_)
{
  real margin = ScalarConvert<accreal, real>::to(margin_);
  THCUNN_assertSameGPU(state, 2, input, target);
  input = THCTensor_(newContiguous)(state, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);
  if (input->dim() == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);
    THCTensor_(resize1d)(state, output, 1);
    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, input->size[0],
        reduction == Reduction::ElementwiseMean,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, input->size[0],
        reduction == Reduction::ElementwiseMean,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->dim() == 2)
  {
    int nframe = input->size[0];
    THArgCheck(!target->is_empty() && (target->dim() == 1) && (target->size[0] == nframe), 3,
               "inconsistent target size");
    dim3 blocks(input->size[0]);
    dim3 threads(MULTIMARGIN_THREADS);

    if (reduction == Reduction::None)
    {
      THCTensor_(resize1d)(state, output, input->size[0]);
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size[1],
          false,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size[1],
          false,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
    }
    else
    {
      THCTensor_(resize1d)(state, output, 1);
      THCTensor *output_ = THCTensor_(newWithSize1d)(state, input->size[0]);  // tmp output buffer
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size[1],
          reduction == Reduction::ElementwiseMean,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          input->size[0], input->size[1],
          reduction == Reduction::ElementwiseMean,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
      float sum = THCTensor_(sumall)(state, output_);
      THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
      THCTensor_(free)(state, output_);
    }
  }
  else
  {
    AT_ERROR("non-empty vector or matrix expected, got sizes: ", input->sizes());
  }

  THCTensor_(free)(state, input);
  if(weights)
    THCTensor_(free)(state, weights);
}

void THNN_(MultiMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction,
           int p,
           THCTensor *weights,
           accreal margin_)
{
  real margin = ScalarConvert<accreal, real>::to(margin_);
  THCUNN_assertSameGPU(state, 3, input, gradInput, target);
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);

  if (input->dim() == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, gradInput->size[0],
        reduction == Reduction::ElementwiseMean,
        margin,
        reduction != Reduction::None
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, gradInput->size[0],
        reduction == Reduction::ElementwiseMean,
        margin,
        reduction != Reduction::None
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->dim() == 2)
  {
    int nframe = gradInput->size[0];
    THArgCheck(!target->is_empty() && (target->dim() == 1) && (target->size[0] == nframe), 3,
               "inconsistent target size");
    dim3 blocks(gradInput->size[0]);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size[1],
        reduction == Reduction::ElementwiseMean,
        margin,
        reduction != Reduction::None
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size[1],
        reduction == Reduction::ElementwiseMean,
        margin,
        reduction != Reduction::None
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else
  {
    AT_ERROR("non-empty vector or matrix expected, got ", input->sizes());
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  if(weights)
    THCTensor_(free)(state, weights);
}

#endif
